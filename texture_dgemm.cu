#include "hip/hip_runtime.h"
#include<stdio.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>
#include "cycleTimer.h"
#define warpSize 32

texture<float4,1> texture_A;
//texture<float,1> texture_B;

__inline__ __device__
float warpReduceSum(float val) {
  for (int offset = warpSize/2; offset > 0; offset /= 2) 
    val += __shfl_down(val, offset);
  return val;
}

__global__
void tileMul(float *_A, float *_B, float *_C,int m, int n, int k, int bm, int bn, int bk, int KtileWidth,int threadPerBlock){
        //assume A is row-major and B is column-major and C has initial value all zero
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	int blockx = blockIdx.x;
	int blocky = blockIdx.y;
	int blockz = blockIdx.z;
	float value[2][2];
	float _value;
	int l,loopi,loopj;
	int tileNum,currentRow;
	int Comb = 4; //float4
	int half = threadPerBlock/2;
	int idy_half = idy-half;
	int idModbm = idy%(bm/2);
	int idDividebm = idy/(bm/2);
    	//int idMod32 = (id%warpSize);
    	//int idDivide32 = (id/warpSize);
	//int subtileN = threadPerBlock/warpSize;
	int totTileNum = min((KtileWidth+bk-1)/bk,(k+bk-1)/bk);
	float *A = _A + blocky * bm*k;
	float *B = _B + blockx * bn*k;
	//float *C = _C + blocky * bm*n + blockx * bn;
	
//	use share memory
	__shared__ float4 tileA[8][16]; //bk*bn
	__shared__ float4 tileB[8][16]; //bk*bn
	__shared__ float tempC[8][8]; //bm*bn,FIXME:dynamic allocate tempC[bn][bm]

	for(tileNum=0 ; tileNum < totTileNum; tileNum++){
		/* tileC(blockx,blocky) += tileA[i] * tileB[i]*/
		int index = blockz*KtileWidth + tileNum*bk + idx*Comb;
		if(index <= k){
            		if(idy<half) tileB[idy][idx] = reinterpret_cast<float4*> (B)[(idy*k + index)/Comb];
					//printf("id=(%d,%d),x=%f,y=%f,z=%f,w=%f,B[%d][%d],offset=%d\n",idx,idy,tileB[idy][idx].x,tileB[idy][idx].y,tileB[idy][idx].z,tileB[idy][idx].w,idy,idx,(idy*k + index)/Comb);}
            		else tileA[idy_half][idx] = reinterpret_cast<float4*> (A)[((idy_half)*k + index)/Comb];
					//printf("id=(%d,%d),x=%f,y=%f,z=%f,w=%f,A[%d][%d],offset=%d\n",idx,idy_half,tileA[idy_half][idx].x,tileA[idy_half][idx].y,tileA[idy_half][idx].z,tileA[idy_half][idx].w,idy_half,idx,(idy_half*k + index)/Comb);}
        	}
		//for(currentRow=0; currentRow < bm; currentRow++){
			//float elementA = A[k*currentRow + blockz*KtileWidth + tileNum*bk + idMod32];//put elementA in reg	
			//for(loop=0; (loop*subtileN)<bn; loop++){
				if(index <= k){
					float4 a0 = tileA[idDividebm*2][idx];
					float4 a1 = tileA[idDividebm*2+1][idx];
					float4 b0 = tileB[idModbm*2][idx];
					float4 b1 = tileB[idModbm*2+1][idx];
					value[0][0] = a0.x*b0.x + a0.y*b0.y + a0.z*b0.z + a0.w*b0.w;
					value[0][1] = a0.x*b1.x + a0.y*b1.y + a0.z*b1.z + a0.w*b1.w;
					value[1][0] = a1.x*b0.x + a1.y*b0.y + a1.z*b0.z + a1.w*b0.w;
					value[1][1] = a1.x*b1.x + a1.y*b1.y + a1.z*b1.z + a1.w*b1.w;
					//printf("index=%d,value=%f,offset=%d\n",index,value,offset);
				}
				else{	
					 value[0][0] = 0;
					 value[0][1] = 0;
					 value[1][0] = 0;
					 value[1][1] = 0;
				}
			/*if(currentRow == 0) printf("Index=%d.block tileNum=%d. Thread %d->%d.Value %f * %f= %f\n",index,tileNum,id,idMod32,elementA, tex1Dfetch(texture_B, (blocky*bn + loop*subtileN + idDivide32)*k + idMod32),value);
			  if(currentRow == 0){
				float valueb = tex1Dfetch(texture_B, (blocky*bn + loop*subtileN + idDivide32)*k + idMod32);
				printf("valueb[%d][%d]=%f\n",blocky * bn+idDivide32+ loop*subtileN, idMod32 ,valueb);
			  }
			*/
        	    //Shuffle Warp Reduce 
		for(loopi=0;loopi<2;loopi++)
			for(loopj=0;loopj<2;loopj++){
				_value=value[loopi][loopj];	
        	    for (l=warpSize/2; l>=2; l/=2)
        	        _value += __shfl_down(_value, l);
	      	    //if(currentRow == 0) printf("Thread %d final value = %f\n", id, value);
        	    if(idx == 0) {
					//tempC[currentRow][idDivide32+loop*32] +=value;		
					tempC[idDividebm*2+loopi][idModbm*2+loopj] += _value;		
					//printf("C_temp[%d][%d] final value = %f\n",currentRow,idDivide32+loop*subtileN,value);		    	
					if(tileNum == totTileNum-1){
						//_C[ (currentRow+blockx*bm)*n + blocky * bn + idDivide32+loop*32] = tempC[currentRow*bn + idDivide32+loop*32];
						atomicAdd(&_C[ (idDividebm*2+loopi+blocky*bm)*n + blockx*bn + idModbm*2+loopj] , tempC[idDividebm*2+loopi][idModbm*2+loopj]);
						//if(currentRow == 0) printf("C[%d][%d] final value = %f\n",currentRow+blockx*bm,idDivide32+loop*32+blocky*bn,tempC[currentRow*bn + idDivide32+loop*32]);
					}
    			}
			}
			//}
		//}
	}
}
int main(int argc, char* argv[]){
	int m,n,k;
	int i,j;
//	m=4096; n=4096; k=4096;
	int microTile = 8;
	sscanf( argv[ 1 ], "%d", &m );
	sscanf( argv[ 2 ], "%d", &n );
	sscanf( argv[ 3 ], "%d", &k );
	sscanf( argv[ 4 ], "%d", &microTile );
	float *A = (float*)malloc(sizeof(float)*m*k);
	float *B = (float*)malloc(sizeof(float)*k*n);
	float *C = (float*)malloc(sizeof(float)*m*n);
//for cublas
/*	for(j=0; j<k; j++){
		for(i=0; i<m; i++){
			A[j*m+i]=(10*i+j)*0.01; //store A in column major, size m*k
		}
	}
*/
//for tile_mul

	for(i=0; i<m; i++){
		for(j=0; j<k; j++){
			A[i*k+j]=i+j*0.01; //store A in row major, size m*k
		}
	}

	for(j=0; j<n; j++){
		for(i=0; i<k; i++){
			B[j*k+i]=i+j*0.01;//store B in column major, size k*n
                }
        }

        for(i=0; i<m; i++){
                for(j=0; j<n; j++){
                        C[i*n+j]=0; //store A in row major, size m*k
                }
        }
/*
	if(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte) != hipSuccess)
		printf("SharedMemBankSizeEightByte failed.\n");
	hipSharedMemConfig pConfig;
	hipDeviceGetSharedMemConfig(&pConfig);
	printf("cudaSharedMemBankSize=%d\n",pConfig);//hipSharedMemBankSizeDefault = 0
						     //hipSharedMemBankSizeFourByte = 1
						     //hipSharedMemBankSizeEightByte = 2
*/
	float* dev_A,*dev_B,*dev_C;
	hipMalloc((void**)&dev_A,m*k*sizeof(float));	
	hipMalloc((void**)&dev_B,k*n*sizeof(float));	
	/*
	size_t pitch;//=warpsize
	hipMallocPitch((void**)&dev_A,&pitch,k*sizeof(float),m);
	hipMallocPitch((void**)&dev_B,&pitch,k*sizeof(float),n);
	*/
	hipMalloc((void**)&dev_C,m*n*sizeof(float));	
	hipMemcpy(dev_A,A,m*k*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_B,B,k*n*sizeof(float),hipMemcpyHostToDevice);
	/*
	hipMemcpy2D(dev_A,pitch,A,sizeof(float)*k,sizeof(float)*k,m,hipMemcpyHostToDevice);
	hipMemcpy2D(dev_B,pitch,B,sizeof(float)*k,sizeof(float)*k,n,hipMemcpyHostToDevice);
	*/
	cublasInit();
	//float alpha = 1.0;
	//float beta = 0.0;
	hipBindTexture(NULL,texture_A,dev_A,m*k*sizeof(float));
	//hipBindTexture(NULL,texture_B,dev_B,k*n*sizeof(float));
	int bm=8;
	int bn=8;//32*4
	int bk=64;
	//int texCache = 48*1024; //48K
	//int microTile = texCache/sizeof(float)/bn/bk;//12
	int threadPerBlock = 16;
	int N = (n+bn-1)/bn;//gridX
	int M = (m+bm-1)/bm;//gridY
	int KtileWidth = bk*microTile;//12*32/2
	int K = (k+KtileWidth-1)/KtileWidth;//gridZ
	dim3 grid_dim(N,M,K);
	dim3 block_dim(threadPerBlock,threadPerBlock,1);
	int blockNum = M*N*K;
	printf("block number %d*%d*%d= %d\n",N,M,K,blockNum);
	double cpuStartTime = CycleTimer::currentSeconds();
	tileMul<<<grid_dim,block_dim>>>(dev_A, dev_B, dev_C, m, n, k,bm,bn,bk,KtileWidth,threadPerBlock);
//	hipblasDgemm( 'n', 'n', m, n, k, alpha, dev_A, m, dev_B, k, beta, dev_C, m);
	hipDeviceSynchronize();
	double cpuEndTime = CycleTimer::currentSeconds();
	double runtime = 1000.f * (cpuEndTime-cpuStartTime);
	double flop = (double)2*m*n*k;
        printf("Dgemm runtime: %.3f ms, GFLOPS=%.6f\n", runtime,flop/runtime/1000000 );
	hipMemcpy(C,dev_C,m*n*sizeof(float),hipMemcpyDeviceToHost);
//	hipUnbindTexture(texture_A);
//	hipUnbindTexture(texture_B);
	cublasShutdown();
	printf("cuda blas:\n");
	printf("m=%d,n=%d,k=%d ",m,n,k);
	for(i=0; i<m; i++){
                printf("\n");
                for(j=0; j<n; j++)
                        printf("%f      ",C[i*n+j]);
        }
		
	return 0;
}

