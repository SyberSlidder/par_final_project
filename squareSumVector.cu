#include "hip/hip_runtime.h"
#include "main.h"

#if __CUDA_ARCH__ >= 300
__device__ inline float warpReduce(float value, int laneID){
    // Use XOR mode to perform butterfly reduction
    #pragma unroll
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor(value, i, 32);
    return value;
}
#else
__device__ inline float warpReduce(float value, int laneID){
    volatile __shared__ float values[1024];
    values[threadIdx.x] = 0.0;
    values[threadIdx.x] = value;
    if(laneID < 16){
        for(int i=16; i>=1; i/=2){
            values[threadIdx.x] += values[threadIdx.x+i];
        }
    }
    
    return values[threadIdx.x];
}
#endif


__global__ void calcSquareSumVector(float *srcMatrix,
                                    float *sqSumVector,
                                    int    M,
                                    int    K){
                                    
    // Shared data
    volatile __shared__ float sdata[32];

    // Calculate thread index and stride
    int laneId = threadIdx.x & 0x1f;
    int icol   = blockIdx.x*blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;
    int warpId = threadIdx.x/32;

    // Thread-Local sum
    float mySqSum = 0.0;

    // Split rows amongst thread blocks
    for(int row  = blockIdx.y;
            row  < M;
            row += gridDim.y){

        // Strided reduction of squared values across columns
        for(int col  = icol;
                col  < K + blockDim.x;
                col += stride){

            // Square the assignmed matrix cell
            float val = (col >= K) ? 0.0 : srcMatrix[K*row + col];
            float sqVal = val*val;

            // Add to thread-local sum
            mySqSum += sqVal;
        }

        // Warp-level reduction with butterfly shuffles
        float warpSqSum = warpReduce(mySqSum,laneId);

        // Store warp-local square-sum
        if(laneId == 0){
            sdata[warpId] = warpSqSum;
        }
        __syncthreads();

        // Lowest work finishes off work
        if(warpId == 0){
            // Read warp-local square-sums
            mySqSum = sdata[laneId];
            //printf("===%3d %3d %3d %5.2f\n", row, warpId, laneId, mySqSum);

            // Add to block-local square sums
            float blkSqSum = warpReduce(mySqSum,laneId);

            // Store result
            if(laneId == 0){
                sqSumVector[row*gridDim.x+blockIdx.x] = blkSqSum;
            }
        }
    }
}
