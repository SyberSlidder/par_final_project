#include "hip/hip_runtime.h"
#include "main.h"


__global__ void MaxwellCombinedSGEMM_v1(
       float * _A, // Global pointer to matrix A 
       float * _B, // Global pointer to matrix B
       float * _C, // Global pointer to write out result of A*B
       float * sqSumVecA, // M x 1 matrix derived from A
       float * sqSumVecB, // N x 1 matrix derived from B
       int M, // Number rows of A
       int N, // Number of columns of B
       int K  // Columns A, rows B
)
{  

    // Complex Grid
    //__shared__ float smA[16][32];
    //__shared__ float smB[16][32];
  
    __shared__ float smA[16][32];
    __shared__ float smB[16][32];
    
    // Initialization
    float cVal[8][8] = { 
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f}
		  };
	
	float track1[8];
	float track2[8];
		  
	// Identification
	int linearThreadID = threadIdx.x + (blockDim.x * threadIdx.y);
	int warpID = linearThreadID / 32;
	
	
	
	// Where to read from in A
	int loadRowA = ((blockDim.y*8) * blockIdx.y) + linearThreadID;
	float * aReadPtr = _A + (loadRowA * K);
	// Where to read from in B
	int loadRowB = (blockDim.y*8) * blockIdx.y;
	float * bReadPtr = _B + (loadRowB * K);
	
	// Loop through the K dimension of Matrices A and B
	// We are operating out of a 64 x 16 chunk of A/B
	for (int i = 0; i < (K/8); i++) {
	    
		// Load from A into SM
		//float4 aHolder1 = *((float4 *)(aReadPtr));
		//float4 aHolder2 = *((float4 *)(aReadPtr + 4));
	  
		//smA[][]
		
		// Load from B into SM
		
		// Update pointers
				
		// Grab 1 8-element track from A and 1 8-elemnt track from B
		// 8 tracks total from A, 8 tracks total from B
		int startIndex = ((linearThreadID & 0x1F) / 16) * 8;
		
		int trackStartID = 0;//(linearThreadID % 8);
		int secondHalf = ((linearThreadID & 0x0F) / 8)*8;
		int trackSelect  = secondHalf + (16 * warpID);
		for (int trackNum = 0; trackNum < 8; trackNum++) {
		
			// Load Track from A into track1
			// Reads down a track
			int columnSelect = trackSelect + trackStartID;
			for (int trackElement = 0; trackElement < 8; trackElement++) {
			    track1[trackElement] = smA[startIndex + trackElement][columnSelect];
			}
			
			for (int trackElement = 0; trackElement < 8; trackElement++) {
			    track2[trackElement] = smB[startIndex + trackElement][columnSelect];
			}
			// Load Track from B into track2
			
			// Compute the outer product of the the tracks
			// 64 FMA from each track 1,2 pair
			// 64 x 8 = 512 FMA operations for loading 64 elements from SM
			#pragma unroll
			for (int trackRow = 0; trackRow < 8; trackRow++) {
				cVal[trackRow][0] += track1[trackRow]*track2[0];
				cVal[trackRow][1] += track1[trackRow]*track2[1];
				cVal[trackRow][2] += track1[trackRow]*track2[2];
				cVal[trackRow][3] += track1[trackRow]*track2[3];
				cVal[trackRow][4] += track1[trackRow]*track2[4];
				cVal[trackRow][5] += track1[trackRow]*track2[5];
				cVal[trackRow][6] += track1[trackRow]*track2[6];
				cVal[trackRow][7] += track1[trackRow]*track2[7];
			}
			
			// Move to the next track
			trackStartID = (trackStartID + 1) % 8;
			
		}
		
	}

	// Write back C
	int C_row = loadRowA;
	int C_column = loadRowB;
	
	for (int i = 0; i < 8; i++) {
	  for (int j = 0; j < 8; j++) {
	    _C[(C_row+i)*N + C_column+j] = cVal[i][j];
	  }
	}
	
}

union dblFloat4 {
  float4 wideFloats[2];
  float  elem[8];
};

__global__ void MaxwellCombinedSGEMM_v2(
       float * _A, // Global pointer to matrix A 
       float * _B, // Global pointer to matrix B
       float * _C, // Global pointer to write out result of A*B
       float * sqSumVecA, // M x 1 matrix derived from A
       float * sqSumVecB, // N x 1 matrix derived from B
       float * _W, // Weight vector
	 int M, // Number rows of A
       int N, // Number of columns of B
       int K  // Columns A, rows B
)
{  

    // Complex Grid
    //__shared__ float smA[16][32];
    //__shared__ float smB[16][32];
  
    __shared__ float smA[16][32];
    __shared__ float smB[16][32];
    
    // Initialization
    float cVal[8][8] = { 
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f},
		  {0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f,0.0f}
		  };
	
	float track1[8];
	float track2[8];
		  
	// Identification
	int linearThreadID = threadIdx.x + (blockDim.x * threadIdx.y);
	int warpID = linearThreadID / 32;
	
	// Where to read from in A
	int loadRowA = ((blockDim.y*8) * blockIdx.y) + linearThreadID;
	float * aReadPtr = _A + (loadRowA * K);
	// Where to read from in B
	int loadRowB = (blockDim.y*8) + (linearThreadID / 8);
	int bColumnOffset = (blockDim.x*blockIdx.x*8) + (8*threadIdx.x);
	float * bReadPtr = _B + (loadRowB * N) + bColumnOffset;
	
	
	// Loop through the K dimension of Matrices A and B
	// We are operating out of a 64 x 16 chunk of A/B
	int columnStart = (linearThreadID >> 3) & 0x03;
	int rowStart = warpID * 8;
	
	dblFloat4 aHolder;
	dblFloat4 bHolder;
	
	for (int i = 0; i < (K/8); i++) {
	    
		// Load from A into register
		aHolder.wideFloats[0] = *((float4 *)(aReadPtr));
		aHolder.wideFloats[1] = *((float4 *)(aReadPtr + 4));
		// Fix the bank and store down a row
		int storeAOffset = linearThreadID % 8;
		for (int trackElement = 0; trackElement < 8; trackElement++) {
		      smA[rowStart + trackElement][linearThreadID % 32] = aHolder.elem[trackElement];
		      //storeAOffset = (storeAOffset + 1 ) % 8;
		}
		
		// Load from B into register
		bHolder.wideFloats[0] = *((float4 *)(bReadPtr));
		bHolder.wideFloats[1] = *((float4 *)(bReadPtr + 4));
		// Fix the bank and store down a row when storing B
		for (int trackElement = 0; trackElement < 8 ; trackElement++) {
			smB[rowStart + trackElement][linearThreadID % 32] = bHolder.elem[trackElement]; 
		}
		
		// Update pointers
		aReadPtr += 8;
		bReadPtr += 8*N;
				
		// Wait for everyone to finish their loads
		__syncthreads();
				
		// Grab 1 8-element track from A and 1 8-elemnt track from B
		// 8 tracks total from A, 8 tracks total from B

		//int offSet = linearThreadID % 8;
		
		for (int trackNum = 0; trackNum < 8; trackNum++) {
		
			// Load Track from A into track1
			// Reads down a track
			
			// Fix row and change banks when reading A from shared memory
			int offSet = linearThreadID % 8;
			for (int trackElement = 0; trackElement < 8; trackElement++) {
			    //track1[trackElement] = smA[rowStart + trackElement][offSet + columnStart];
			    track1[offSet] = smA[rowStart + trackNum][columnStart + offSet];
			    offSet = (offSet + 1) % 8;
			}
			
			for (int trackElement = 0; trackElement < 8; trackElement++) {
			    track2[trackElement] = smB[rowStart + trackElement][offSet + columnStart];
			}
			// Load Track from B into track2
			
			// Compute the outer product of the the tracks
			// 64 FMA from each track 1,2 pair
			// 64 x 8 = 512 FMA operations for loading 64 elements from SM
			#pragma unroll
			for (int trackRow = 0; trackRow < 8; trackRow++) {
				cVal[trackRow][0] += track1[trackRow]*track2[0];
				cVal[trackRow][1] += track1[trackRow]*track2[1];
				cVal[trackRow][2] += track1[trackRow]*track2[2];
				cVal[trackRow][3] += track1[trackRow]*track2[3];
				cVal[trackRow][4] += track1[trackRow]*track2[4];
				cVal[trackRow][5] += track1[trackRow]*track2[5];
				cVal[trackRow][6] += track1[trackRow]*track2[6];
				cVal[trackRow][7] += track1[trackRow]*track2[7];
			}
			
			// Move to the next track
			//offSet = (offSet + 1) % 8;
			
		}
		
	}

	// Write back C
	int C_row = loadRowA;
	int C_column = loadRowB;
	
	if (GEMM_ONLY) {
		for (int i = 0; i < 8; i++) {
			for (int j = 0; j < 8; j++) {
				_C[(C_row+i)*N + C_column+j] = cVal[i][j];
			}
		}
	} else {
		// Load 8 elements from the weight vector
		float weights[8];
		float * weightPtr = _W + C_column;
		float4 w1,w2;
		w1 = (*(float4 *)(weightPtr));
		w2 = (*(float4 *)(weightPtr+4)); 		
		
		
		
		
	}
	
}