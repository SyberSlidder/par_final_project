#include "hip/hip_runtime.h"
#include "smem_ops.h"
#include "equations.h"

namespace inplace {
namespace detail {

//Work around nvcc/clang bug on OS X
#ifndef __clang__

template __global__ void smem_row_shuffle(int m, int n, float* d, c2r::shuffle s);
template __global__ void smem_row_shuffle(int m, int n, double* d, c2r::shuffle s);

template __global__ void smem_row_shuffle(int m, int n, int* d, c2r::shuffle s);
template __global__ void smem_row_shuffle(int m, int n, long long* d, c2r::shuffle s);

template __global__ void smem_row_shuffle(int m, int n, float* d, r2c::shuffle s);
template __global__ void smem_row_shuffle(int m, int n, double* d, r2c::shuffle s);

template __global__ void smem_row_shuffle(int m, int n, int* d, r2c::shuffle s);
template __global__ void smem_row_shuffle(int m, int n, long long* d, r2c::shuffle s);

#else
namespace {

template<typename A, typename B>
void* magic() {
    return (void*)&smem_row_shuffle<A, B>;
}


template void* magic<float, c2r::shuffle>();
template void* magic<double, c2r::shuffle>();
template void* magic<int, c2r::shuffle>();
template void* magic<long long, c2r::shuffle>();

template void* magic<float, r2c::shuffle>();
template void* magic<double, r2c::shuffle>();
template void* magic<int, r2c::shuffle>();
template void* magic<long long, r2c::shuffle>();

}

#endif

}
}
